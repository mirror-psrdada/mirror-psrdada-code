#include "ipcutil_cuda.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <errno.h>
#include <string.h>

#include <sys/ipc.h>
#include <sys/sem.h>
#include <sys/shm.h>

// #define _DEBUG 1

/* *************************************************************** */
/*!
  Returns a device pointer to a shared memory block and its shmid
*/
void* ipc_alloc_cuda (key_t key, size_t size, int flag, int* shmid, void ** shm_addr, int device_id)
{
  void * devPtr = 0;
  hipIpcMemHandle_t * handlePtr = 0;
  int id = 0;
  size_t handle_size = sizeof(hipIpcMemHandle_t);
  hipError_t error;

#ifdef _DEBUG
  fprintf (stderr, "ipc_alloc_cuda: shmget(key=%x size=%ld, flag=%x)\n",
           key, handle_size, flag);
#endif

  // we want to extract the IPC handle
  id = shmget (key, handle_size, flag);
  if (id < 0) 
  {
     fprintf (stderr, "ipc_alloc_cuda: shmget (key=%x, size=%ld, flag=%x) %s\n",
              key, handle_size, flag, strerror(errno));
     return 0;
  }

#ifdef _DEBUG
  fprintf (stderr, "ipc_alloc_cuda: shmid=%d\n", id);
#endif

  // pointer to hipIpcMemHandle_t
  *shm_addr = shmat (id, 0, flag);
  if (*shm_addr == (void *)-1) 
  {
    fprintf (stderr,
       "ipc_alloc_cuda: shmat (shmid=%d) %s\n"
       "ipc_alloc_cuda: after shmget (key=%x, size=%ld, flag=%x)\n",
       id, strerror(errno), key, size, flag);
    return 0;
  }

  handlePtr = (hipIpcMemHandle_t *) *shm_addr;

#ifdef _DEBUG
  fprintf (stderr, "ipc_alloc_cuda: shm_addr=%p handlePtr=%p\n", *shm_addr, handlePtr);
  fprintf (stderr, "ipc_alloc_cuda: selecting device %d\n", device_id);
#endif

  error = hipSetDevice (device_id);
  if (error != hipSuccess)
  {
    fprintf (stderr, "failed to select cuda device %s: %s\n",
             device_id, hipGetErrorString (error));
    return 0;
  }

#ifdef _DEBUG
  fprintf (stderr, "ipc_alloc_cuda: selected device %d\n", device_id);
#endif
  // if we are wanting to create a shared memory segment of size bytes
  if (flag & IPC_CREAT)
  {
    // allocate device memory
#ifdef _DEBUG
    fprintf (stderr, "ipc_alloc_cuda: allocating device memory of size %ld bytes\n", size);
#endif
    error = hipMalloc (&devPtr, size);
    if (error != hipSuccess)
    {
      fprintf (stderr, "failed to allocate %ld bytes on device %d: %s\n", 
               size, device_id, hipGetErrorString (error));
      return 0;
    }
#ifdef _DEBUG
    fprintf (stderr, "ipc_alloc_cuda: hipIpcGetMemHandle (%p, %p)\n", handlePtr, devPtr);
#endif

    // get an event handle associated with that memory, writing to handlePtr
    error = hipIpcGetMemHandle (handlePtr, devPtr);
    if (error != hipSuccess)
    {
      fprintf (stderr, "failed to get IPC memory handle for devPtr=%p on device %d: %s\n", 
               devPtr, device_id, hipGetErrorString (error));
      return 0;
    }
  }
  else
  {
#ifdef _DEBUG
    fprintf (stderr, "ipc_alloc_cuda: hipIpcOpenMemHandle()\n");
#endif
    // get a pointer to existing device memory using handlePtr
    error = hipIpcOpenMemHandle (&devPtr, *handlePtr, hipIpcMemLazyEnablePeerAccess);
    if (error != hipSuccess)
    {
      fprintf (stderr, "ipc_alloc_cuda: failed to open memory handle to segment: %s\n",
              hipGetErrorString (error));
      return 0;
    }
  }

#ifdef _DEBUG
  fprintf (stderr, "ipc_alloc_cuda: devPtr=%p\n", devPtr);
#endif

  if (shmid)
    *shmid = id;

  return devPtr;
}

// detach from memory segment
int ipc_disconnect_cuda (void * devPtr)
{
#ifdef _DEBUG
  fprintf (stderr, "ipc_disconnect_cuda: ipc_disconnect_cuda(%p)\n", devPtr);
#endif
  hipError_t error = hipIpcCloseMemHandle (devPtr);
  if (error != hipSuccess)
  {
    fprintf (stderr, "ipc_disconnect_cuda: failed to close memory handle to segment: %s\n",
             hipGetErrorString (error));
    return -1;
  }

  // now restore 
  return 0;
}

// deallocate memory segment
int ipc_dealloc_cuda (void * devPtr, int device_id)
{
  hipSetDevice (device_id);
#ifdef _DEBUG
  fprintf (stderr, "ipc_disconnect_cuda: hipFree(%p)\n", devPtr);
#endif
  hipError_t error = hipFree (devPtr);
  if (error != hipSuccess)
  {
    fprintf (stderr, "ipc_disconnect_cuda: failed to free device memory: %s\n",
             hipGetErrorString (error));
    return -1;
  }

  return 0;
}

int ipc_zero_buffer_cuda (void * devPtr, size_t nbytes)
{
  hipError_t error = hipMemset (devPtr, 0, nbytes);
  if (error != hipSuccess)
  {
    fprintf (stderr, "ipc_zero_buffer_cuda: failed to zero device memory: %s\n",
             hipGetErrorString (error));
    return -1;
  }

  return 0;
}

